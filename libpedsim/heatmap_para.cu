#include "hip/hip_runtime.h"
// Created for Low Level Parallel Programming 2017
//
// Implements the heatmap functionality. 
//
#include "ped_model.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
#include "hip/hip_runtime.h"
#include ""
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

// TODO:
// 1. Something must use shared memory
// 2. Maybe move desiredX/desiredY to memory, but would this give speedup or the contrary?

// Sets up the heatmap
void Ped::Model::setupHeatmapPara()
{
	hipMallocManaged(&hm, SIZE*SIZE*sizeof(int));
	hipMallocManaged(&shm, SCALED_SIZE*SCALED_SIZE*sizeof(int));
	hipMallocManaged(&bhm, SCALED_SIZE*SCALED_SIZE*sizeof(int));

	// These are just an abstraction to those above, using pointers
	heatmap = (int**)malloc(SIZE*sizeof(int*));
	scaled_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
	blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

	for (int i = 0; i < SIZE; i++)
	{
		heatmap[i] = hm + SIZE*i;
	}
	for (int i = 0; i < SCALED_SIZE; i++)
	{
		scaled_heatmap[i] = shm + SCALED_SIZE*i;
		blurred_heatmap[i] = bhm + SCALED_SIZE*i;
	}

	// This replaces the calloc call
	for (int i = 0; i < SIZE * SIZE; i++)
	{
		hm[i] = 0;
	}
	/*
	for (int i = 0; i < agents.size(); i++)
	{
		agents_desiredX[i] = agents[i]->getDesiredX();
		agents_desiredY[i] = agents[i]->getDesiredY();
	}
	*/

}

__global__
void fadeOutAgentsKernel(int *heatmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	heatmap[x+y*SIZE] = (int)round(heatmap[x+y*SIZE] * 0.80);
}

__global__
void paintHeatmap(int *heatmap, int *blurred_heatmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;// - blockIdx.x * 2;
	int y = blockIdx.y * blockDim.y + threadIdx.y;// - blockIdx.y * 2;
	int const blockSize = 16;
	//int tidx = (threadIdx.x - blockIdx.x * 2) % blockSize;
	//int tidy = (threadIdx.y - blockIdx.y * 2) % blockSize;
	int tidx = threadIdx.x + 2;
	int tidy = threadIdx.y + 2;

	if (x > SCALED_SIZE || y > SCALED_SIZE)
		return;


	// Get values from the heatmap
	int const cellSize = 5;
	int hmX = x / cellSize; 
	int hmY = (y / cellSize) * SIZE;
	int value = heatmap[hmX+hmY];
	
	// The scaled heatmap - using shared memory
	int const padding = 4;
	__shared__ int sh[padding + blockSize][padding + blockSize]; // we need padding for gaussian blur
	sh[tidx][tidy] = value;
	
	// TODO: We also need the values of the padding

	// Weights for blur filter
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};

#define WEIGHTSUM 273
	// Apply gaussian blurfilter	
	int i = x + 2;
	int j = y + 2;

	if (tidx < 2 || tidx >= 14 || tidy < 2 || tidy >= 14)
		return;
	if (i >= SCALED_SIZE - 2 || j >= SCALED_SIZE - 2)
		return; 

	int sum = 0;
	for (int k = -2; k < 3; k++)
	{
		for (int l = -2; l < 3; l++)
		{
			sum += w[2 + k][2 + l] * sh[tidy + k][tidx + l];
		}
	}
	value = sum / WEIGHTSUM;
	blurred_heatmap[(j * SCALED_SIZE) + i] = 0x00FF0000 | value << 24;
}


// Updates the heatmap according to the agent positions
void Ped::Model::fadeOutAgents()
{
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(SIZE / threadsPerBlock.x, SIZE / threadsPerBlock.y);
	fadeOutAgentsKernel<<<numBlocks, threadsPerBlock>>>(hm);
}


void Ped::Model::updateHeatmapPara()
{
	hipDeviceSynchronize();

	// Count how many agents want to go to each location
	#pragma parallel for
	for (int i = 0; i < agents.size(); i++)
	{
		Ped::Tagent* agent = agents[i];
		int x = agent->getDesiredX();
		int y = agent->getDesiredY();

		if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
		{
			continue;
		}

		// intensify heat for better color results
		int hm_value = hm[x+y*SIZE];
		hm[x+y*SIZE] = (hm_value < (255 - 40)) ? hm_value + 40 : 255;

	}
	// Paint the heatmap
	dim3 threadsPerBlock(16, 16);
	// num blocks = SCALED_SIZE / threadsPerBlock.x + SCALED_SIZE / (threadsPerBlock.x - 2) + SCALED_SIZE / threadsPerBlock.x 
	dim3 numBlocks((SCALED_SIZE) / threadsPerBlock.x, (SCALED_SIZE) / threadsPerBlock.y);
	paintHeatmap<<<numBlocks,threadsPerBlock>>>(hm, bhm);
	hipDeviceSynchronize();
}

#include "hip/hip_runtime.h"
// Created for Low Level Parallel Programming 2017
//
// Implements the heatmap functionality. 
//
#include "ped_model.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

// Sets up the heatmap
void Ped::Model::setupHeatmapPara()
{
	// Scaled heatmap (shm) is actually not used, but has to be
	// initsialized to avoid seg faults.
	hipMallocManaged(&hm, SIZE*SIZE*sizeof(int));
	hipMallocManaged(&shm, SCALED_SIZE*SCALED_SIZE*sizeof(int));
	hipMallocManaged(&bhm, SCALED_SIZE*SCALED_SIZE*sizeof(int));

	// These are just an abstraction to those above, using pointers
	heatmap = (int**)malloc(SIZE*sizeof(int*));
	scaled_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
	blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

	for (int i = 0; i < SIZE; i++)
	{
		heatmap[i] = hm + SIZE*i;
	}
	for (int i = 0; i < SCALED_SIZE; i++)
	{
		scaled_heatmap[i] = shm + SCALED_SIZE*i;
		blurred_heatmap[i] = bhm + SCALED_SIZE*i;
	}

	// This replaces the calloc call
	for (int i = 0; i < SIZE * SIZE; i++)
	{
		hm[i] = 0;
	}
}

__global__
void fadeOutAgentsKernel(int *heatmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	heatmap[x+y*SIZE] = (int)round(heatmap[x+y*SIZE] * 0.80);
}

__global__
void paintHeatmap(int *heatmap, int *blurred_heatmap)
{
	// Since the gaussian blur are working the area around the pixles and
	// needs 2 pixles of padding, this will cause trouble when we're using a 
	// scaled heatmap in shared memory that only contains the pixles of the current
	// block. This means we have to put padding in each block - essentially this means
	// that each calculated block only moves BLOCKSIZE-2 at a time. That is, we must calculate
	// some extra blocks.
	int const padding = 4;
	int x = blockIdx.x * blockDim.x + threadIdx.x - padding * blockIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y - padding * blockIdx.y;
	int tidx = threadIdx.x;
	int tidy = threadIdx.y;

	if (x > SCALED_SIZE || y > SCALED_SIZE)
		return;

	// Get values from the heatmap
	int const cellSize = 5;
	int hmX = x / cellSize;
	int hmY = (y / cellSize) * SIZE;
	int value = heatmap[hmX+hmY];
	
	// The scaled heatmap - using shared memory
	int const blockSize = 16;
	__shared__ int sh[blockSize][blockSize]; // we need padding for gaussian blur
	sh[tidy][tidx] = value;

	// Weights for blur filter
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};


	// These pixles are used as padding so ignore these
	if (tidx < 2 || tidy < 2 || tidx >= (blockSize - 2) || tidy >= (blockSize - 2))
		return;

	// Apply gaussian blurfilter	
#define WEIGHTSUM 273
	int sum = 0;
	for (int k = -2; k < 3; k++)
	{
		for (int l = -2; l < 3; l++)
		{
			sum += w[2 + k][2 + l] * sh[tidy + k][tidx + l];
		}
	}
	value = sum / WEIGHTSUM;
	blurred_heatmap[(y * SCALED_SIZE) + x] = 0x00FF0000 | value << 24;
}


// Updates the heatmap according to the agent positions
void Ped::Model::fadeOutAgents()
{
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(SIZE / threadsPerBlock.x, SIZE / threadsPerBlock.y);
	fadeOutAgentsKernel<<<numBlocks, threadsPerBlock>>>(hm);
}


void Ped::Model::updateHeatmapPara()
{
	hipDeviceSynchronize();

	// Count how many agents want to go to each location
	#pragma omp parallel for
	for (int i = 0; i < agents.size(); i++)
	{
		Ped::Tagent* agent = agents[i];
		int x = agent->getDesiredX();
		int y = agent->getDesiredY();

		if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
		{
			continue;
		}

		// intensify heat for better color results
		int hm_value = hm[x+y*SIZE];
		hm[x+y*SIZE] = (hm_value < (255 - 40)) ? hm_value + 40 : 255;
	}

	// Paint the heatmap
	const int offset = 2; // some offset is used for the gaussian blur padding
	const int bs = 16; // blocksize

	// Setup and start kernel
	dim3 threadsPerBlock(bs, bs);
	dim3 numBlocks(
		(SCALED_SIZE) / (threadsPerBlock.x - offset), 
		(SCALED_SIZE) / (threadsPerBlock.y - offset));
	paintHeatmap<<<numBlocks,threadsPerBlock>>>(hm, bhm);
	hipDeviceSynchronize();
}
